#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <ctime>
// Includes CUDA
#include <hip/hip_runtime.h>

#define LINEWIDTH 20
#define NWORDS 32
#define N_STREAMS 8
#define BLOCK_SIZE 32
#define TITLE_SIZE 1

int length;
int len;
int nwords;
int matches[NWORDS];
char *ctext;
char keywords[NWORDS][LINEWIDTH];
unsigned int  *text;
unsigned int  *words;
float cpuRunTime;

// citation: https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void intialise(char *input)
{
    nwords = NWORDS;
    printf("-----------\nGoint to read %s\n", input);
    char *line;
	line = (char*) malloc(sizeof(char)*LINEWIDTH);
	memset(matches, 0, sizeof(matches));


	// read in text and keywords for processing
	FILE *fp, *wfile;
	wfile = fopen("./data/keywords.txt","r");
	if (!wfile)
	{	printf("keywords.txt: File not found.\n");	exit(0);}

	int k=0, cnt = nwords;
	size_t read, linelen = LINEWIDTH;
	while((read = getline(&line, &linelen, wfile)) != -1 && cnt--)
	{
		strncpy(keywords[k], line, sizeof(line));
		keywords[k][4] = '\0';
		k++;
	}
	fclose(wfile);


	fp = fopen(input,"r");
	if (!fp)
	{	printf("Unable to open the file.\n");	exit(0);}

	length = 0;
	while (getc(fp) != EOF) length++;
	ctext = (char *)malloc(length+4);

	rewind(fp);

	for (int l=0; l<length; l++) ctext[l] = getc(fp);
	for (int l=length; l<length+4; l++) ctext[l] = ' ';

	fclose(fp);

	printf("Length : %d\n", length );
	// define number of words of text, and set pointers
	len  = length/4;
	text = (unsigned int *) ctext;

	// define words for matching
	words = (unsigned int *)malloc(nwords*sizeof(unsigned int));

	for (int w=0; w<nwords; w++)
	{
		words[w] = ((unsigned int) keywords[w][0])
             + ((unsigned int) keywords[w][1])*(1<<8)
             + ((unsigned int) keywords[w][2])*(1<<16)
             + ((unsigned int) keywords[w][3])*(1<<24);

	}
}

void deinit(){
	free(words);
    free(text);
}

void check_matches(int *temp_matches){
	bool isRight = true;
    for(int i = 0; i<NWORDS; i++) {
		if(matches[i] != temp_matches[i]) {
			isRight = false;
			printf("WRONG OUTPUT:\t %s\t|\t%d\n",  keywords[i], temp_matches[i]);
        }
	}

	if(isRight) {
		printf(" - Correct Answer -\n");
	}
}

void print_matches(int *temp_matches){
    printf("Printing Matches:\n");
	printf("Word\t  |\tNumber of Matches\n===================================\n");
	for (int i = 0; i < nwords; ++i)
		printf("%s\t  |\t%d\n", keywords[i], temp_matches[i]);

}

void matchPattern_CPU(unsigned int *text, unsigned int *words, int *matches, int nwords, int length)
{
	unsigned int word;

	for (int l=0; l<length; l++)
	{
		for (int offset=0; offset<4; offset++)
		{
			if (offset==0)
				word = text[l];
			else
				word = (text[l]>>(8*offset)) + (text[l+1]<<(32-8*offset)); 

			for (int w=0; w<nwords; w++){
				matches[w] += (word==words[w]);
			} 
				
		}
	}
}

void exec_CPU(){
    // CPU execution
	const clock_t begin_time = clock();
	matchPattern_CPU(text, words, matches, nwords, len);
	cpuRunTime = (float)( clock() - begin_time ) /  CLOCKS_PER_SEC;
	printf("CPU exec time: %f s\n\n", cpuRunTime);
}

__global__ void matchPattern_gpu_1(unsigned int *text, unsigned int *words, int *matches, int nwords, int length, int offset_, int which)
{
	int tid = threadIdx.x;
	int idx = offset_ + blockIdx.x * blockDim.x + tid;
	
	// for loading text into the shared memory
	__shared__ unsigned int text_s[NWORDS + 1];
	text_s[tid] = text[idx];
	text_s[NWORDS] = text[offset_ + (blockIdx.x * blockDim.x) + blockDim.x];

	// loads the keyword for this thread
	// each thread in a block is reponsible for one keyword
	unsigned int keyword = words[tid];
	
	__syncthreads();
	
	unsigned int word;
	int sum = 0;
 

		
	#pragma loop unroll
	for(int w = 0; w < NWORDS; w++) {
		#pragma loop unroll
		for (int offset=0; offset<4; offset++)
		{
			word = offset==0 ? text_s[w] : (text_s[w]>>(8*offset)) + (text_s[w+1]<<(32-8*offset)); 
			sum = sum + (word==keyword);
		}
	}

	atomicAdd(&matches[(which*NWORDS)+tid],sum);
}

// citation: https://github.com/NVIDIA-developer-blog/code-samples/blob/master/series/cuda-cpp/overlap-data-transfers/async.cu
void exec_gpu_stream(){

	const int nStreams = N_STREAMS;
	const int streamSize = len / nStreams;
	const int straming_bytes = streamSize * sizeof(unsigned int);

	unsigned int *d_text; unsigned int *d_words; int *d_matches;
	int *h_matches;
	h_matches = (int *)malloc(nwords*sizeof(int)*N_STREAMS);
	memset(h_matches, 0, nwords*sizeof(int)*N_STREAMS);

	hipStream_t stream[nStreams];
	for (int i = 0; i < nStreams; i++){
		checkCudaErrors( hipStreamCreate(&stream[i]));
	}

	hipHostRegister(words,nwords*sizeof(int),0);
	hipHostRegister(text,strlen(ctext)*sizeof(char),0);
	hipHostRegister(h_matches,nwords*sizeof(int),0);

	checkCudaErrors(hipMalloc((void**)&d_words, nwords*sizeof(unsigned int)));
	checkCudaErrors(hipMalloc((void**)&d_matches, nwords*sizeof(int)*nStreams));
	checkCudaErrors(hipMalloc((void**)&d_text, sizeof(unsigned int)*len));
		
	hipEvent_t start, stop;	
	float tiime_ = 0;
	checkCudaErrors( hipEventCreate(&start) );
	checkCudaErrors( hipEventCreate(&stop) );

	checkCudaErrors( hipEventRecord(start,0) );
	checkCudaErrors(hipMemcpy(d_words, words, nwords*sizeof(unsigned int), hipMemcpyHostToDevice));
	for (int i = 0; i < nStreams; ++i) 
	{
		int offset = i * streamSize;
		checkCudaErrors(hipMemcpyAsync(&d_text[offset], &text[offset], straming_bytes, hipMemcpyHostToDevice, stream[i]));
	}
	
	for (int i = 0; i < nStreams; ++i) 
	{
		int offset = i * streamSize;
		matchPattern_gpu_1<<<ceil(streamSize/(TITLE_SIZE*NWORDS)), NWORDS, 0, stream[i]>>>(d_text, d_words, d_matches, nwords, len, offset, i);
	}

	for (int i = 0; i < nStreams; ++i) 
	{
		int offset = i * streamSize;
		checkCudaErrors(hipMemcpyAsync(&h_matches[(i*NWORDS)], &d_matches[(i*NWORDS)], NWORDS*sizeof(int), hipMemcpyDeviceToHost, stream[i]));
	}
	// cudaMemcpy(h_matches, d_matches, nwords*sizeof(int), cudaMemcpyDeviceToHost);
	checkCudaErrors( hipEventRecord(stop, 0) );
	checkCudaErrors( hipEventSynchronize(stop) );
	checkCudaErrors( hipEventElapsedTime(&tiime_, start, stop) );
	  
	printf("Time kernel+memory: %fs\n", tiime_/1000);
	printf("Speedup with memory: %f\n", cpuRunTime/((tiime_)/1000));

	for(int w=0; w < NWORDS; w++)
	{
		for (int i = 1; i < nStreams; ++i)
		{
			h_matches[w] += h_matches[(i*NWORDS) + w];
		}
	}

	check_matches(h_matches);

	// cleanup
	checkCudaErrors( hipEventDestroy(start) );
	checkCudaErrors( hipEventDestroy(stop) );

	for (int i = 0; i < nStreams; ++i) {
		checkCudaErrors( hipStreamDestroy(stream[i]));
	}
	
	hipHostUnregister(text);
	hipHostUnregister(words);
	hipHostUnregister(h_matches);

	hipFree(d_words);
	hipFree(d_matches);
	hipFree(d_text);
	
}
void exec_gpu_simple(){

		// GPU execution
		unsigned int *d_text; unsigned int *d_words; int *d_matches;
		int *h_matches;
		h_matches = (int *)malloc(nwords*sizeof(int));


		checkCudaErrors(hipMalloc((void**)&d_words, nwords*sizeof(unsigned int)));
		checkCudaErrors(hipMalloc((void**)&d_matches, nwords*sizeof(int)));
		checkCudaErrors(hipMalloc((void**)&d_text, sizeof(unsigned int)*len));

		hipEvent_t start,stop;
		float time_H2D,time_D2H,time_kernel;
		checkCudaErrors(hipEventCreate(&start));
		checkCudaErrors(hipEventCreate(&stop));


		// MEMCOPY
		hipEventRecord(start, 0);
		checkCudaErrors(hipMemcpy(d_words, words, nwords*sizeof(unsigned int), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_text, text, sizeof(unsigned int)*len, hipMemcpyHostToDevice));
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_H2D,start,stop);
		printf("HostToDevice memcopy time: %fs\n", time_H2D/1000);

		// RUN KERNEL
		hipEventRecord(start, 0);
		matchPattern_gpu_1<<< ceil((float)len/(TITLE_SIZE*NWORDS)),NWORDS>>>(d_text, d_words, d_matches, nwords, len, 0, 0);
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		checkCudaErrors(hipPeekAtLastError());
		hipEventElapsedTime(&time_kernel,start,stop);
		printf("Kernel execution time: %fs\n", time_kernel/1000);
		
		hipEventRecord(start, 0);
		checkCudaErrors(hipMemcpy(h_matches, d_matches, nwords*sizeof(int), hipMemcpyDeviceToHost));
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_D2H,start,stop);
		printf("DeviceToHost memcopy time: %fs\n", time_D2H/1000);

		printf("Total memcopy time: %fs\n", (time_D2H+time_H2D)/1000);
		printf("Total memcopy+kernel time: %fs\n", (time_D2H+time_H2D+time_kernel)/1000);
		
		printf("Speedup without memory: %f\n", cpuRunTime/((time_kernel)/1000));
		printf("Speedup with memory: %f\n", cpuRunTime/((time_D2H + time_H2D + time_kernel)/1000));
		
		check_matches(h_matches);

		hipEventDestroy(start);
		hipEventDestroy(stop);

	    free(h_matches);
		hipFree(d_words);
		hipFree(d_matches);
		hipFree(d_text);
}

int main(int argc, const char **argv)
{

	intialise("./data/small.txt");
    exec_CPU();
	exec_gpu_stream();

	intialise("./data/medium.txt");
    exec_CPU();
	exec_gpu_stream();

    intialise("./data/large.txt");
    exec_CPU();
	exec_gpu_stream();

	deinit();
	
}
